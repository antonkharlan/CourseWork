#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <chrono>
#include <string>
#include <windows.h>
#include <intrin.h>

using namespace std;

#define blockX 32
#define blockY 16

__global__ void kernelGPU(int* a, int* b, int rows, int cols) {

	int x = blockIdx.x * blockDim.x;
	//blockDim – размер блока; blockIdx – индекс текущего блока в сетке;
	int y = blockIdx.y * blockDim.y;
	int i = threadIdx.x + x;
	//threadIdx – индекс текущей нити в блоке;
	int j = threadIdx.y + y;
	if ((i < rows) && (j < cols)) {
		b[i * cols + j] = a[j * rows + i];
	}
}

//Сравнение матриц
int compareMatrix(int* a, int* b, int rows, int cols) {
	for (int i = 0; i < rows * cols; i++) {
		if (a[i] != b[i]) {
			cout << "	!" << endl;
			return 1;
		}
	}
	cout << "Матрицы GPU и CPU равны!" << endl;
	return 0;
}

//Трансконирование матрицы
int CPUfunction(int* a, int* b, int rows, int cols) {
	for (int i = 0; i < rows; i++) {
		for (int j = 0; j < cols; j++) {
			b[i * cols + j] = a[j * rows + i];
		}
	}
	return 0;
}

//Вывод информации об устройстве
static void deviceInfo() {
	cout << "Информация об используемых устройствах:" << endl;
	//CPU
	int CPUInfo[4] = { -1 };
	__cpuid(CPUInfo, 0x80000000);
	unsigned int nExIds = CPUInfo[0];

	char CPUBrandString[0x40] = { 0 };
	for (unsigned int i = 0x80000000; i <= nExIds; ++i)
	{
		__cpuid(CPUInfo, i);

		if (i == 0x80000002)
		{
			memcpy(CPUBrandString,
				CPUInfo,
				sizeof(CPUInfo));
		}
		else if (i == 0x80000003)
		{
			memcpy(CPUBrandString + 16,
				CPUInfo,
				sizeof(CPUInfo));
		}
		else if (i == 0x80000004)
		{
			memcpy(CPUBrandString + 32, CPUInfo, sizeof(CPUInfo));
		}
	}
	cout << "Процессор: " << CPUBrandString << endl;
	SYSTEM_INFO siSysInfo;
	GetSystemInfo(&siSysInfo);
	cout << "Количество ядер: " << siSysInfo.dwNumberOfProcessors << endl;

	//GPU
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	for (int i = 0; i < deviceCount; ++i)
	{
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, i);
		cout << "Видеокарта: " << deviceProp.name << endl;
		cout << "Полная глобальная память: " << deviceProp.totalGlobalMem << " байт" << endl;
		cout << "Максимальное количество потоков на блок: " << deviceProp.maxThreadsPerBlock << endl;
		cout << "Максимальное количество потоков на мультипроцессор: " << deviceProp.maxThreadsPerMultiProcessor << endl;
		cout << "Максимальные размеры сетки: " << deviceProp.maxGridSize[0] << " x " << deviceProp.maxGridSize[1] << " x " << deviceProp.maxGridSize[2] << endl;
		cout << "Максимальные размеры блока: " << deviceProp.maxThreadsDim[0] << " x " << deviceProp.maxThreadsDim[1] << " x " << deviceProp.maxThreadsDim[2] << endl;
		//cout << "Общая память на блок: " << deviceProp.sharedMemPerBlock << endl;
	}

}

// Вывод созданных и используемых в программе матриц для проверки  
void showMatrix(int* a, int choose, int rows, int cols) {
	if (choose == 1) {
		//Вывод стартовой матрицы на экран(10x10)
		for (int i = 0; i < 10; i++) {
			for (int j = 0; j < 10; j++) {
				cout << setw(2) << a[j * rows + i] << " ";
			}
			cout << endl;
		}
	}
	else {
		//Вывод транспонированных матриц на экран(10x10)
		for (int i = 0; i < 10; i++) {
			for (int j = 0; j < 10; j++) {
				cout << setw(2) << a[j * cols + i] << " ";
			}
			cout << endl;
		}
	}
	cout << endl;
}

static void Info(int rows, int cols) {
	cout << "--------------------------------" << endl;
	cout << "Текущий размер матрицы:" << rows << "x" << cols << endl;
}


int main() {

	setlocale(LC_ALL, "Rus");
	
	int cols = 10;
	int rows = 10;

	int* hostA, * hostB, * hostC;
	int* devA, * devB;
	
	//Создание массива(таблицы) зависимости времени от размера матрицы
	string output[8][4];
	output[0][0] = { "Размер матрицы:    " };
	output[0][1] = { "Время CPU:         " };
	output[0][2] = { "Время GPU:         " };
	output[0][3] = { "Отношение СPU/GPU: " };

	for (int k = 1; k <= 7;) {

		//Выделение памяти 
		hostA = (int*)malloc(sizeof(int) * cols * rows);
		hostB = (int*)malloc(sizeof(int) * cols * rows);
		hostC = (int*)malloc(sizeof(int) * cols * rows);

		for (int i = 0; i < cols; i++) {	
			for (int j = 0; j < rows; j++) {
					hostA[i * rows + j] = (int)rand() % 100;
			}
		}

		// Выделение памяти на GPU.
		hipMalloc(&devA, sizeof(int) * rows * cols);
		hipMalloc(&devB, sizeof(int) * rows * cols);

		// Скопировать входные данные из памяти CPU в память GPU.
		hipMemcpy(devA, hostA, sizeof(int) * rows * cols, hipMemcpyHostToDevice);

		// Задать конфигурацию запуска n нитей.
		dim3 blockDim = dim3(blockX, blockY);
		dim3 gridDim = dim3((rows + 31) / 32, (cols + 15) / 16);

		// Объявление переменных-событий начала и окончания
		// Используем cudaEvent для замеров времени
		hipEvent_t gpuStart, gpuStop;

		// Инициализация переменных-событий.
		hipEventCreate(&gpuStart);
		hipEventCreate(&gpuStop);
		// Привязка события start к данной позиции в коде
		// программы (начало выполнения ядра).
		hipEventRecord(gpuStart, 0);

		// Запуск GPU-ядра.
		kernelGPU << < gridDim, blockDim >> > (devA, devB, rows, cols);
		// Привязка события stop к данной позиции в коде
		// программы (окончание выполнения ядра).
		hipEventRecord(gpuStop, 0);
		// Ожидание окончания выполнения ядра,
		// синхронизация по событию stop.
		hipEventSynchronize(gpuStop);
		float elapsedTime;
		// Получение времени, прошедшего между событиями start и stop.
		hipEventElapsedTime(&elapsedTime, gpuStart, gpuStop);
		
		//Вызов вывода текущего состояния
		Info(rows, cols);

		cout << "GPU time: " << elapsedTime << "ms" << endl;
		
		// Скопировать результаты в память CPU.
		hipMemcpy(hostB, devB, sizeof(int) * cols * rows, hipMemcpyDeviceToHost);

		//Повторение вычислений на процессоре
		auto start = std::chrono::high_resolution_clock::now();

		CPUfunction(hostA, hostC, rows, cols);

		auto stop = std::chrono::high_resolution_clock::now();
		std::chrono::duration<double> time = stop - start;
		
		cout << "CPU time: " << time.count() * 1000 << "ms" << endl;

		//Формирование таблицы вывода результатов
		string ratio = to_string(time.count() * 1000 / elapsedTime);
		output[k][0] = {"| " + to_string(rows) + "x" + to_string(cols) + " "};
		output[k][1] = {"| " + to_string(time.count() * 1000) + "ms " };
		output[k][2] = {"| " + to_string(elapsedTime) + "ms " };
		output[k][3] = {"| " + ratio + " " };

		
		cout << endl << "StartMatrix:" << endl;
		showMatrix(hostA, 1, rows, cols);
		cout << "GPU:" << endl;
		showMatrix(hostB, 2, rows, cols);
		cout << "CPU:" << endl;
		showMatrix(hostC, 2, rows, cols);
		

		//Сравнение матриц
		compareMatrix(hostB, hostC, rows, cols);

		//Очистка созданной динамической памяти
		free(hostA);
		free(hostB);
		free(hostC);
		// Освободить выделенную память GPU.
		hipFree(devA);
		hipFree(devB);

		k++;

		//Задание размера матриц 10, 50, 100, 500, 1000, 5000, 10000
		if (k % 2 == 0) {
			cols *= 5;
			rows *= 5;
		}
		else {
			cols = cols * 2;
			rows = rows * 2;
		}
	}
	cout << endl;

	deviceInfo();

	//Вывод таблицы на экран
	cout << endl;
	for (int i = 0; i < 4; i++) {
		for (int j = 0; j < 8; j++) {
			cout << output[j][i] << "\t";
		}
		cout << endl;
	}
	return 0;
}